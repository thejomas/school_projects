#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h> 

#include "transpose-kernels.cu.h"
#include "transpose-host.cu.h"

#define HEIGHT_A 1024*8   //12835//2048//2048
#define  WIDTH_A 1024*8  //15953 //1024//2048
#define TILE     32
#define RUNS_GPU 100

int gpuAssert(hipError_t code) {
  if(code != hipSuccess) {
    printf("GPU Error: %s\n", hipGetErrorString(code));
    return -1;
  }
  return 0;
}

int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1)
{
    unsigned int resolution=1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec);
    result->tv_sec = diff / resolution;
    result->tv_usec = diff % resolution;
    return (diff<0);
}


/**
 * Measure a more-realistic optimal bandwidth by a simple, memcpy-like kernel
 */ 
int bandwidthMemcpy( const uint32_t B     // desired CUDA block size ( <= 1024, multiple of 32)
                   , const size_t   N     // length of the input array
                   , float* d_in          // device input  of length N
                   , float* d_out         // device result of length N
) {
    // dry run to exercise the d_out allocation!
    const uint32_t num_blocks = (N + B - 1) / B;
    naiveMemcpy<<< num_blocks, B >>>(d_out, d_in, N);
    
    double gigaBytesPerSec;
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    { // timing the GPU implementations
        gettimeofday(&t_start, NULL); 

        for(int i=0; i<RUNS_GPU; i++) {
            naiveMemcpy<<< num_blocks, B >>>(d_out, d_in, N);
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
        gigaBytesPerSec = 2 * N * sizeof(int) * 1.0e-3f / elapsed;
        printf("Naive Memcpy GPU Kernel runs in: %lu microsecs, GB/sec: %.2f\n\n\n"
              , elapsed, gigaBytesPerSec);
    }
 
    gpuAssert( hipPeekAtLastError() );
    return 0;
}

void randomInit(float* data, int size) {
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}


template<class T>
void matTranspose(T* A, T* trA, int rowsA, int colsA) {
  for(int i = 0; i < rowsA; i++) {
    for(int j = 0; j < colsA; j++) {
      trA[j*rowsA + i] = A[i*colsA + j];
    }
  }
}

template<class T>
bool validateTranspose(float* A,float* trA, unsigned int rowsA, unsigned int colsA){
  bool valid = true;
  for(unsigned int i = 0; i < rowsA; i++) {
    for(unsigned int j = 0; j < colsA; j++) {
      if(trA[j*rowsA + i] != A[i*colsA + j]) {
        printf("row: %d, col: %d, A: %.4f, trA: %.4f\n", 
                i, j, A[i*colsA + j], trA[j*rowsA + i] );
        valid = false;
        break;
      }
    }
    if(!valid) break;
  }
  if (valid) printf("GPU TRANSPOSITION   VALID!\n");
  else       printf("GPU TRANSPOSITION INVALID!\n");
  return valid;
}


bool validateProgram(float* A, float* B, unsigned int N){
  bool valid = true;
  for(unsigned int i = 0; i < N; i++) {
    unsigned long long ii = i*64;
    double accum = 0.0;
    for(int j = 0; j < 64; j++) {
        float tmpA  = A[ii + j];
        accum = sqrt(accum) + tmpA*tmpA;
        if(fabs(B[ii+j] - accum) > 0.00001) {
            printf("Row %d column: %d, seq: %f, par: %f\n", i, j, accum, B[ii+j]);
            valid = false; break; 
        }
    }
    if(!valid) break;
  }
  if (valid) printf("GPU PROGRAM   VALID!\n");
  else       printf("GPU PROGRAM INVALID!\n");
  return valid;
}


int weekly3Task3( int height
                , float* h_A
                , float* h_B
                , float* d_A
                , float* d_B
) {
    const uint32_t width = 64; // each row has 64 float elements
    const size_t mem_size = height * width * sizeof(float);
    const unsigned int REPEAT = RUNS_GPU;
    double gigaBytesPerSec;

    { // compute original program
        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL); 

        unsigned int block      = 256;
        unsigned int num_thds   = height;
        unsigned int num_blocks = (height + block - 1) / block;

        for (int kkk = 0; kkk < REPEAT; kkk++) {
            origProg<<<num_blocks, block>>>(d_A, d_B, num_thds);
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / REPEAT; 
        gigaBytesPerSec = 2 * mem_size * 1.0e-3f / elapsed;
        printf( "Original Program runs on GPU in: %lu microsecs, GB/sec: %f\n"
              , elapsed, gigaBytesPerSec);

        // copy result from device to host
        hipMemcpy(h_B, d_B, mem_size, hipMemcpyDeviceToHost);
        gpuAssert( hipPeekAtLastError() );
        hipMemset(d_B, 0, mem_size);
        validateProgram(h_A, h_B, num_thds);
    }

    { // Compute transformed program in which all read and write 
      // accesses are coalesced. This is obtained by transposing
      // input array A and result array B.

        // We allocate device buffers for d_Atr---i.e., the transposed of A,
        // and for d_Btr---i.e., the transpose of B
        float* d_Atr;   hipMalloc((void**) &d_Atr, mem_size);
        float* d_Btr;   hipMalloc((void**) &d_Btr, mem_size);
 
        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL); 

        unsigned int num_thds   = height;
        unsigned int block      = 256;
        unsigned int num_blocks = (num_thds + block - 1) / block;

        // Task3.a) ToDo: 
        //       - fill in the implementation CPU orchestration code below
        //       - and the corresponding CUDA kernel!
        for (int kkk = 0; kkk < REPEAT; kkk++) {
            // 3.a.1  you probably need to transpose d_A here by
            //        using function "transposeTiled<float, TILE>"
            //        i.e., source array is d_A, result array is d_Atr
            // 3.a.2  you probably need to implement the "transfProg"
            //        kernel in file transpose-kernel.cu.h which takes 
            //        input from d_Atr, and writes the result in d_Btr,
            transfProg<<< num_blocks, block >>>(d_Atr, d_Btr, num_thds);
            // 3.a.3  you probably need to transpose-back the result here
            //        i.e., source array is d_Btr, and transposed result
            //        is in d_B.
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / REPEAT; 
        gigaBytesPerSec = 2 * mem_size * 1.0e-3f / elapsed;
        printf( "Coalesced Program with manifested transposition runs on GPU in: %lu microsecs, GB/sec: %f\n"
              , elapsed, gigaBytesPerSec);

        // copy result from device to host
        hipMemcpy(h_B, d_B, mem_size, hipMemcpyDeviceToHost);
        gpuAssert( hipPeekAtLastError() );
        hipMemset(d_B, 0, mem_size);

        validateProgram(h_A, h_B, num_thds);

        // deallocate the transposed buffers here
        hipFree(d_Atr);
        hipFree(d_Btr);
   }

    { // Optimized program---i.e., exhibiting only coalesced 
      // accesses---obtained by using the shared memory as
      // a staging buffer, i.e., read from global-to-shared
      // memory (in coalesced way) and then each thread reads
      // from shared memory in non-coalesced way. Note that
      // this version should be the fastest, as it does not
      // require to perform (manifest) the transpositions
      // (in global memory).
      // 
      // Task 3.b) implement function "glb2shmem" in file
      //           "transpose-kernels.cu.h"
        hipMemset(d_B, 0, mem_size);

        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL); 

        unsigned int num_thds= height;
        unsigned int block   = 256;
        unsigned int grid    = (num_thds + block - 1) / block;
        const int CHUNK = 16;

        if((block % CHUNK) != 0) {
            printf("Broken Assumption: block size not a multiple of chunk size, EXITING!\n");
            exit(1);
        }

        for (int kkk = 0; kkk < REPEAT; kkk++) {
            optimProg<CHUNK><<<grid, block, CHUNK*block*sizeof(float)>>>(d_A, d_B, num_thds);
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / REPEAT; 
        gigaBytesPerSec = 2 * mem_size * 1.0e-3f / elapsed;
        printf("Optimized Program runs on GPU in: %lu microsecs, GB/sec: %f\n", elapsed, gigaBytesPerSec);

        gpuAssert( hipPeekAtLastError() );

        // copy result from device to host
        hipMemcpy(h_B, d_B, mem_size, hipMemcpyDeviceToHost);
        hipMemset(d_B, 0, mem_size);
        validateProgram(h_A, h_B, num_thds);
    }
    return 0;
}


int main() {
    // set seed for rand()
    srand(2006);
 
    // 1. allocate host memory for the two matrices
    size_t size_A = WIDTH_A * HEIGHT_A;
    size_t mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    float* h_B = (float*) malloc(mem_size_A);
 
    // 2. initialize host memory
    randomInit(h_A, size_A);
    
    // 3. allocate device memory
    float* d_A;
    float* d_B;
    hipMalloc((void**) &d_A, mem_size_A);
    hipMalloc((void**) &d_B, mem_size_A);
 
    // 4. copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

    bandwidthMemcpy( 256, size_A, d_A, d_B );
    gpuAssert( hipPeekAtLastError() );

    { // test transpose
        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL); 

        for (int kkk = 0; kkk < RUNS_GPU; kkk++) {
            //transposeNaive<float, TILE>( d_A, d_B, HEIGHT_A, WIDTH_A );
            transposeTiled<float, TILE>( d_A, d_B, HEIGHT_A, WIDTH_A );
        }
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU; 
        double gigaBytesPerSec = 2.0 * mem_size_A * 1.0e-3f / elapsed;
        printf("Transpose on GPU runs in: %lu microsecs, GB/sec: %f\n", elapsed, gigaBytesPerSec);

        // copy result from device to host
        hipMemcpy(h_B, d_B, mem_size_A, hipMemcpyDeviceToHost);
  
        // validate
        validateTranspose<float>( h_A, h_B, HEIGHT_A, WIDTH_A );
    }
    gpuAssert( hipPeekAtLastError() );

    weekly3Task3( WIDTH_A * HEIGHT_A / 64, h_A, h_B, d_A, d_B );

   // clean up memory
   free(h_A);
   free(h_B);
   hipFree(d_A);
   hipFree(d_B);
}

