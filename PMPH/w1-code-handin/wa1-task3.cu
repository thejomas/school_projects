#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>

int timeval_subtract( struct timeval* result,
                      struct timeval* t2,struct timeval* t1) {
    unsigned int resolution=1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) -
        (t1->tv_usec + resolution * t1->tv_sec) ;
    result->tv_sec = diff / resolution; result->tv_usec = diff % resolution;
    return (diff<0);
}

__global__ void task3Kernel(float* d_in, float *d_out, int N) {
    const unsigned int lid = threadIdx.x; // local id inside a block
    const unsigned int gid = blockIdx.x*blockDim.x + lid; // global id
    if (gid < N) {
        float val = d_in[gid];
        val = val/(val-2.3);
        d_out[gid] = val*val*val; // do computation
    }
}

int seqMap(float *d_in, float *d_out, int N) {
    for (int i=0; i < N; i++) {
        float val = d_in[i];
        val = val/(val-2.3);
        d_out[i] = val*val*val;
    }
    return 0;
}

#define GPU_RUNS 100
#define EPSILON 0.0001
int main(int argc, char** argv) {
    // Shit used to setup the CUDA run
    unsigned int N = 1000000;
    unsigned int mem_size = N*sizeof(float);
    unsigned int block_size = 256;
    unsigned int num_blocks = (N+(block_size - 1))/block_size;

	// allocate host memory
	float* h_in = (float*) malloc(mem_size);
	float* seq_h_out = (float*) malloc(mem_size);
	float* par_h_out = (float*) malloc(mem_size);

	// initialize the memory
	for(unsigned int i=0; i<N; ++i){
		h_in[i] = (float)i;
	}

	// allocate device memory
	float* d_in;
	float* d_out;
	hipMalloc((void**)&d_in, mem_size);
	hipMalloc((void**)&d_out, mem_size);

	// copy host memory to device
	hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    // Setup timers
    unsigned long int elapsed; struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    printf("Running parallel map...\n");
	// execute the kernel
	for (int i=0; i < GPU_RUNS; i++) {
        task3Kernel<<< num_blocks, block_size>>>(d_in, d_out, N);
    } hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / GPU_RUNS;
    printf("Took %d microseconds (%.2fms)\n", elapsed, elapsed/1000.0);

	// copy result from ddevice to host
	hipMemcpy(par_h_out, d_out, mem_size, hipMemcpyDeviceToHost);

	// print result
	// for(unsigned int i=0; i<N; ++i) printf("%.6f\n", par_h_out[i]);

    printf("Running sequential map...\n");
    for (int i=0; i < GPU_RUNS; i++){
        seqMap(h_in, seq_h_out, N);
    } hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / GPU_RUNS;
    printf("Took %d microseconds (%.2fms)\n", elapsed, elapsed/1000.0);
    // for(unsigned int i=0; i<N; ++i) printf("%.6f\n", seq_h_out[i]);

    int eq_res = 1;
    for (int i=0; i < N; i++) {
        if (abs(seq_h_out - par_h_out) < EPSILON){//Def eps
            eq_res=0;
        }
    }
    if (eq_res)
        printf("Same results, yasss\n");
    else
        printf("Different results, noooo\n");

	// clean-up memory
	free(h_in); free(par_h_out); free(seq_h_out);
	hipFree(d_in); hipFree(d_out);
}

/* Copy when you wanna time shit
 hipDeviceSynchronize();
    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / GPU_RUNS;
    printf("Took %d microseconds (%.2fms)\n",elapsed,elapsed/1000.0);
}
*/
